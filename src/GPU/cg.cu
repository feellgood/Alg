#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include <type_traits>

#include "../alg_utils.h"
#include "gpu_utils.h"


/**
I,J indices of the non zero coefficients of the sparse matrix
val : values of the non zero coefficients of the sparse matrix 
x : result of dimension N
rhs : right hand size
N : dimension of the linear system to solve
nz : nb of coefficients of the sparse matrix
tol : tolerance
max_iter : nb iteration maximum
nb_iter : number of iteration done
returns residue
*/

template <class T>
T _cg(alg::CSR_mat<T> const& A, T *x, T *rhs, const T tol, const int max_iter, int &nb_iter)
{
hipDataType size_float;

if (std::is_same<T,float>::value)
	{ size_float = HIP_R_32F; }
else if (std::is_same<T,double>::value)
	{ size_float = HIP_R_64F; }
else exit(1);

const int nz = A.I[A.N];
const int N = A.N;

int *d_col, *d_row;
int k;
T *d_val, *d_x, *d_r, *d_p, *d_Ax;
T r0,r1,a,na,b,dot;

hipblasHandle_t cublasHandle = 0;
hipblasCreate(&cublasHandle);

hipsparseHandle_t cusparseHandle = 0;
hipsparseCreate(&cusparseHandle);

hipMalloc((void **)&d_col, nz * sizeof(int));
hipMalloc((void **)&d_row, (N + 1) * sizeof(int));
hipMalloc((void **)&d_val, nz * sizeof(T));

hipMalloc((void **)&d_x, N * sizeof(T));
hipMalloc((void **)&d_r, N * sizeof(T));
hipMalloc((void **)&d_p, N * sizeof(T));
hipMalloc((void **)&d_Ax, N * sizeof(T));

  // Wrap raw data into cuSPARSE generic API objects
hipsparseSpMatDescr_t matA;
hipsparseCreateCsr(&matA, N, N, nz, d_row, d_col, d_val,HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_BASE_ZERO, size_float);

hipsparseDnVecDescr_t vecx;
hipsparseCreateDnVec(&vecx, N, d_x, size_float);

hipsparseDnVecDescr_t vecp;
hipsparseCreateDnVec(&vecp, N, d_p, size_float);

hipsparseDnVecDescr_t vecAx;
hipsparseCreateDnVec(&vecAx, N, d_Ax, size_float);

hipMemcpy(d_col, A.J, nz * sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(d_row, A.I, (N + 1) * sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(d_val, A.val, nz * sizeof(T), hipMemcpyHostToDevice);
hipMemcpy(d_x,x,N*sizeof(T), hipMemcpyHostToDevice);
hipMemcpy(d_r,rhs,N*sizeof(T), hipMemcpyHostToDevice);

T alpha = 1.0;
T alpham1 = -1.0;
T beta = 0.0;
r0 = 0.0;

// Allocate workspace for cuSPARSE 
size_t bufferSize = 0;
hipsparseSpMV_bufferSize(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecx,&beta, vecAx, size_float, HIPSPARSE_CSRMV_ALG1, &bufferSize);
void *buffer = NULL;
hipMalloc(&buffer, bufferSize);

hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecx, &beta, vecAx, size_float,HIPSPARSE_CSRMV_ALG1, buffer);

cublas_axpy(cublasHandle, N, &alpham1, d_Ax, 1, d_r, 1);
cublas_dot(cublasHandle, N, d_r , 1, d_r, 1, &r1);

k=1;

while(r1 > tol*tol && k <= max_iter)
	{
	if (k>1)
		{
		b = r1/r0;
		cublas_scal(cublasHandle, N, &b, d_p, 1);
		cublas_axpy(cublasHandle, N, &alpha, d_r, 1, d_p, 1);
		}
	else
		{ cublas_copy(cublasHandle, N, d_r, 1, d_p, 1); }
	
	hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecp, &beta, vecAx, size_float, HIPSPARSE_CSRMV_ALG1, buffer);//HIPSPARSE_SPMV_ALG_DEFAULT
	
	cublas_dot(cublasHandle, N, d_p, 1, d_Ax, 1, &dot);
	a = r1/dot;
	
	cublas_axpy(cublasHandle,N,&a,d_p,1,d_x,1);
	na = -a;
	cublas_axpy(cublasHandle,N,&na,d_Ax,1,d_r,1);
	
	r0 = r1;
	cublas_dot(cublasHandle,N,d_r,1,d_r,1,&r1);
	hipDeviceSynchronize();
	k++;
	}

hipMemcpy(x, d_x, N*sizeof(T), hipMemcpyDeviceToHost);

hipsparseDestroy(cusparseHandle);
hipblasDestroy(cublasHandle);

hipsparseDestroySpMat(matA);
hipsparseDestroyDnVec(vecx);
hipsparseDestroyDnVec(vecAx);

hipFree(d_col);
hipFree(d_row);
hipFree(d_val);
hipFree(d_x);
hipFree(d_r);
hipFree(d_p);
hipFree(d_Ax);

nb_iter = k;
return sqrt(r1);
}

double cg(alg::CSR_mat<double> const& A, double *x, double *rhs, const double tol, const int max_iter, int &nb_iter)
{ return _cg<double>(A, x, rhs, tol, max_iter, nb_iter); }

float cg(alg::CSR_mat<float> const& A, float *x, float *rhs, const float tol, const int max_iter, int &nb_iter)
{ return _cg<float>(A, x, rhs, tol, max_iter, nb_iter); }

