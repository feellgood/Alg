#include <iostream>
#include <hip/hip_runtime.h>

void infos(void)
	{ hipDeviceProp_t devProp;
	if (hipSuccess == hipGetDeviceProperties(&devProp,0))
	std::cout << "CUDA Device = " << devProp.major << "." << devProp.minor << " has "<< devProp.multiProcessorCount << " multi-processors" << std::endl;
	}
