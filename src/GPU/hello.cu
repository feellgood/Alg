
#include <hip/hip_runtime.h>
#include<stdio.h>
#include <iostream>

__global__ void hello_kernel(void)
{ 
printf("Hello World! (from GPU)\n");// we have to speak C here (GPU)
}

int main(void)
{
std::cout << "Hello World! (from host)\n"; // here (host) we can use C++
hello_kernel<<<1,1>>>();

hipDeviceProp_t devProp;
if (hipSuccess == hipGetDeviceProperties(&devProp,0))
	std::cout << "CUDA Device = " << devProp.major << "." << devProp.minor << "has "<< devProp.multiProcessorCount << "Multi-processors" << std::endl;

//wait for device to finish to see the result (here the 'hello world' from the GPU)
hipDeviceSynchronize();
std::cout <<"CUDA error: " << hipGetErrorString(hipGetLastError()) << std::endl;
return 0;
}
