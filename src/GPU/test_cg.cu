#include "cg.h"

#include<iostream>
#include<iomanip>

int main(void)
{
const int N =10;
int nz = (N-2)*3 + 4;
int *I, *J ;
float *val, *x, *rhs;

I = new int[N+1];
J = new int[nz];
val = new float[nz];

genTridiag<float>(I, J, val, N, nz);
//for(int i=0;i<nz;i++)	{std::cout<< "val[" << i << "]: " << val[i] <<std::endl;}

x = new float[N];
rhs = new float[N];

for (int i = 0; i<N; i++)
	{
	x[i] = i/10.f;
	rhs[i] = 1.0;
	std::cout<< "x[" << i << "]: " << x[i] <<std::endl;
	}

float tol = 1e-3;
int max_iter = 100;

cg<float>(I,J,val,x,rhs,N,nz,tol,max_iter);

for(int i=0;i<N;i++)
	{std::cout<< "x[" << i << "]: " << x[i] <<std::endl;}
delete[] x;

delete[] I;
delete[] J;
delete[] val;
delete[] rhs;

std::cout <<"CUDA error: " << hipGetErrorString(hipGetLastError()) << std::endl;
return 0;
}
